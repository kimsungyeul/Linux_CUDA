#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <limits.h>                       /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h>

#include <iostream>

#include "bmpHeader.h"

/* 🕔미지 데🕔터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)

typedef unsigned char ubyte;

//Cuda kernel for converting RGB image into a GreyScale image

__global__ void convertToGrey(ubyte *rgb, ubyte *grey, int rows, int cols, int elemSize) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	// Compute for only those threads which map directly to image grid
	if (col < cols && row < rows) {
		int grey_offset = row * cols + col;
		int rgb_offset = grey_offset * elemSize;

		ubyte r = rgb[rgb_offset + 2];
		ubyte g = rgb[rgb_offset + 1];
		ubyte b = rgb[rgb_offset + 0];

		grey[grey_offset] = r * 0.299f + g * 0.587f + b * 0.114f;
	}
}

int main(int argc, char** argv)
{
	FILE* fp;
	BITMAPFILEHEADER bmpHeader;              /* BMP FILE INFO */
	BITMAPINFOHEADER bmpInfoHeader;     /* BMP IMAGE INFO */
	RGBQUAD *palrgb;

	ubyte *inimg, *outimg;

	if(argc != 3) {
		fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
		return -1;
	}

	/***** read bmp *****/
	if((fp=fopen(argv[1], "rb")) == NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}

	/* BITMAPFILEHEADER 구조체의 데🕔터 */
	fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);

	/* BITMAPINFOHEADER 구조체의 데🕔터 */
	fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);

	/* 트루 컬러를 지원하면 변환할 수 없다. */
	if(bmpInfoHeader.biBitCount != 24) {
		perror("This image file doesn't supports 24bit color\n");
		fclose(fp);
		return -1;
	}

	int elemSize = bmpInfoHeader.biBitCount/8.;
	int stride = bmpInfoHeader.biWidth * elemSize;

	//widthbytes(bits) (((bits)+31)/32*4)

	int imageSize = stride * bmpInfoHeader.biHeight;
	int graySize = bmpInfoHeader.biWidth*bmpInfoHeader.biHeight;

	/* 🕔미지의 해상도(넓🕔 × 깊🕔) */

	printf("Resolution : %d x %d\n", bmpInfoHeader.biWidth, bmpInfoHeader.biHeight);
	printf("Bit Count : %d(%d:%d)\n", bmpInfoHeader.biBitCount, elemSize, stride);     /* 픽셀당 비트수(색상) */
	printf("Image Size : %d\n", imageSize);

	inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
	outimg = (ubyte*)malloc(sizeof(ubyte)*graySize);
	fread(inimg, sizeof(ubyte), imageSize, fp);

	fclose(fp);

#if 0
	for(int y = 0; y < bmpInfoHeader.biHeight; y++) {
		for(int x = 0; x < stride; x+=elemSize) {
			ubyte b = inimg[x+y*stride+0];
			ubyte g = inimg[x+y*stride+1];
			ubyte r = inimg[x+y*stride+2];
			
			//outimg[x/elemSize+y*bmpInfoHeader.biWidth]=((66*r+129*g+25*b+128)>>8)+16;
			outimg[x/elemSize+y*bmpInfoHeader.biWidth]=(r*0.299F)+(g*0.587F)+(b*0.114F);
		}
	}
#else
	ubyte *d_inimg = NULL, *d_outimg = NULL;

	//allocate and initialize memory on device
	hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
	hipMalloc(&d_outimg, sizeof(ubyte) * graySize);
	hipMemset(d_outimg, 0, sizeof(ubyte) * graySize);

	//copy host rgb data array to device rgb data array
	hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/16));
	const dim3 dimBlock(32, 16);

	//execute cuda kernel
	convertToGrey<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);

	//copy computed gray data array from device to host
	hipMemcpy(outimg, d_outimg, sizeof(ubyte) * graySize, hipMemcpyDeviceToHost);

	hipFree(d_outimg);
	hipFree(d_inimg);
#endif

	/***** write bmp *****/
	if((fp=fopen(argv[2], "wb"))==NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}

	palrgb = (RGBQUAD*)malloc(sizeof(RGBQUAD)*256);
	
	for(int x = 0; x < 256; x++) {
		palrgb[x].rgbBlue = palrgb[x].rgbGreen = palrgb[x].rgbRed = x;
		palrgb[x].rgbReserved = 0;
	}

	bmpInfoHeader.biBitCount = 8;
	bmpInfoHeader.SizeImage = imageSize;

	bmpInfoHeader.biCompression = 0;
	bmpInfoHeader.biClrUsed = 0;
	bmpInfoHeader.biClrImportant = 0;

	bmpHeader.bfOffBits = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + sizeof(RGBQUAD)*256;
	bmpHeader.bfSize = bmpHeader.bfOffBits + bmpInfoHeader.SizeImage;

	/* BITMAPFILEHEADER 구조체의 데🕔터 */
	fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);

	/* BITMAPINFOHEADER 구조체의 데🕔터 */
	fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	fwrite(palrgb, sizeof(RGBQUAD), 256, fp);
	//fwrite(inimg, sizeof(ubyte), imageSize, fp);
	fwrite(outimg, sizeof(ubyte), graySize, fp);

	fclose(fp);

	free(inimg);
	free(outimg);

	return 0;
}

