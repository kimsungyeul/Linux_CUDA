#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
int main()
{
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	
	for (int i=0; i< count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		printf(prop.deviceOverlap?"Enabled\n":"Disabled\n");
		printf("Kernel execu1on timeout : " );
		printf(prop.kernelExecTimeoutEnabled?"Enabled\n":"Disabled\n");
		printf("\n" );
		printf(" --- Memory Informa1on for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf("\n");
		printf(" --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],
		prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0],
		prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");	
	}
	
	return 0;
}
