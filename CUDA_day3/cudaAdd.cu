
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *sum){
	*sum = *a + *b;
}

int main(int argc, char** argv)
{
	int a = 2, b = 4, sum;
	int *dev_a,*dev_b,*dev_sum;
	int size = sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_sum, size);

	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);

	add<<<1, 1>>>(dev_a, dev_b, dev_sum);

	hipMemcpy(&sum, dev_sum, size, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_sum);

	printf("2 + 4 = %d from CUDA\n", sum);

	return 0;
}
