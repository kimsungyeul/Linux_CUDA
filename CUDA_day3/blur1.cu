#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <string.h>
#include <limits.h>                     /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h> 

#include "bmpHeader.h"

/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)

typedef unsigned char ubyte;

//Cuda kernel for converting RGB image into a GreyScale image
__global__ void convertBlur(ubyte *in, ubyte *out, int rows, int cols, int elemSize) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    
    // define the kernel
    float kernel[3][3] = { {1/9.0, 1/9.0, 1/9.0},
                           {1/9.0, 1/9.0, 1/9.0},
                           {1/9.0, 1/9.0, 1/9.0} };

    int offset = (x + y * cols) * elemSize;
	for(z = 0; z < elemSize; z++) {
        if((x > 0 && x < (cols - 1)) && ((y > 0) && (y < (rows-1)))) {
            float sum = 0.0;
            for(int i = -1; i < 2; i++) {
                for(int j = -1; j < 2; j++) {
                    //sum += in[((x-i)+(y-j)*cols)*elemSize+z];
		    sum += kernel[i+1][j+1]*in[((x+i)+(y+j)*cols)*elemSize+z];
                }
            }
            //out[offset+z] = sum/9.;
            out[offset+z] = sum;
        } else if (y == 0) {
            float sum = 0.0;
            ubyte arr[9];
    	if(x == 0) {
            arr[0] = arr[1] = arr[3] = arr[4] = in[x*elemSize+z];
	        arr[2] = arr[5] = in[(x+1)*elemSize+z];
    	    arr[6] = arr[7] = in[(x+(y+1)*cols)*elemSize+z];
    	    arr[8] = in[((x+1)+(y+1)*cols)*elemSize+z];
    	} else if(x == (cols-1)) {
            arr[1] = arr[2] = arr[4] = arr[5] = in[x*elemSize+z];
	        arr[0] = arr[3] = in[(x-1)*elemSize+z];
	        arr[6] = in[((x-1)+(y+1)*cols)*elemSize+z];
	        arr[7] = arr[8] = in[(x+(y+1)*cols)*elemSize+z];
        } else {
            arr[3] = arr[0] = in[(x-1)*elemSize+z];
            arr[4] = arr[1] = in[x*elemSize+z];
            arr[5] = arr[2] = in[(x+1)*elemSize+z];
    	    arr[6] = in[((x-1)+(y+1)*cols)*elemSize+z];
    	    arr[7] = in[(x+(y+1)*cols)*elemSize+z];
    	    arr[8] = in[((x+1)+(y+1)*cols)*elemSize+z];
    	}

	    for(int i = 0; i < 9; i++)
    	    sum += arr[i];
            out[offset+z] = sum/9;
        } else if (y == (rows - 1)) {
            float sum = 0.0;
            ubyte arr[9];
            if(x == 0) {
                arr[6] = arr[7] = \
	                arr[3] = arr[4] = in[(x  +(y  )*cols)*elemSize+z];
	            arr[8] = arr[5] = in[(x+1+(y  )*cols)*elemSize+z];
    	        arr[0] = arr[1] = in[(x  +(y-1)*cols)*elemSize+z];
	            arr[2] = in[((x+1)+(y-1)*cols)*elemSize+z];
	        } else if(x == (cols-1)) {
                arr[7] = arr[8] = \
                    arr[4] = arr[5] = in[(x  +(y  )*cols)*elemSize+z];
	            arr[6] = arr[3] = in[(x-1+(y  )*cols)*elemSize+z];
    	        arr[1] = arr[2] = in[(x  +(y-1)*cols)*elemSize+z];
	            arr[0] = in[((x-1)+(y-1)*cols)*elemSize+z];
            } else {
                arr[3] = arr[6] = in[(x-1+y*cols)*elemSize+z];
                arr[4] = arr[7] = in[(x  +y*cols)*elemSize+z];
                arr[5] = arr[8] = in[(x+1+y*cols)*elemSize+z];
                arr[0] = in[((x-1)+(y-1)*cols)*elemSize+z];
	            arr[1] = in[((x  )+(y-1)*cols)*elemSize+z];
    	        arr[2] = in[((x+1)+(y-1)*cols)*elemSize+z];
            }

            for(int i = 0; i < 9; i++)
	            sum += arr[i];
            out[offset+z] = sum/9;
        } else if (x == 0) {
            float sum = 0.0;
            ubyte arr[9];
            arr[0] = arr[1] = in[(x+(y-1)*cols)*elemSize+z];
            arr[3] = arr[4] = in[(x+(y  )*cols)*elemSize+z];
            arr[6] = arr[7] = in[(x+(y+1)*cols)*elemSize+z];
            arr[2] = in[((x+1)+(y-1)*cols)*elemSize+z];
            arr[5] = in[((x+1)+(y  )*cols)*elemSize+z];
            arr[8] = in[((x+1)+(y+1)*cols)*elemSize+z];
            for(int i = 0; i < 9; i++)
	            sum += arr[i];
            out[offset+z] = sum/9;
        } else if (x == (cols-1)) {
            float sum = 0.0;
            ubyte arr[9];
            arr[1] = arr[2] = in[(x+(y-1)*cols)*elemSize+z];
            arr[4] = arr[5] = in[(x+(y  )*cols)*elemSize+z];
            arr[7] = arr[8] = in[(x+(y+1)*cols)*elemSize+z];
            arr[0] = in[((x-1)+(y-1)*cols)*elemSize+z];
            arr[3] = in[((x-1)+(y  )*cols)*elemSize+z];
            arr[6] = in[((x-1)+(y+1)*cols)*elemSize+z];
            for(int i = 0; i < 9; i++)
	            sum += arr[i];
            out[offset+z] = sum/9;
        }            
    }
}

int main(int argc, char** argv) 
{
    FILE* fp; 
    BITMAPFILEHEADER bmpHeader;             	/* BMP FILE INFO */
    BITMAPINFOHEADER bmpInfoHeader;     	/* BMP IMAGE INFO */
    ubyte *inimg, *outimg;

    if(argc != 3) {
        fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
        return -1;
    }
    
    
    if((fp = fopen(argv[1], "rb")) == NULL) { 		/* read bmp */ 
        fprintf(stderr, "Error : Failed to open file...₩n"); 
        return -1;
    }

    /* BITMAPFILEHEADER 구조체의 데이터 */
    fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);

    /* BITMAPINFOHEADER 구조체의 데이터 */
    fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);

    /* 트루 컬러를 지원하면 변환할 수 없다. */
    if(bmpInfoHeader.biBitCount != 24) {
        perror("This image file doesn't supports 24bit color\n");
        fclose(fp);
        return -1;
    }
    
    int elemSize = bmpInfoHeader.biBitCount/8.;
    int stride = bmpInfoHeader.biWidth * elemSize;
    //widthbytes(bits) (((bits)+31)/32*4)
    int imageSize = stride * bmpInfoHeader.biHeight; 

    /* 이미지의 해상도(넓이 × 깊이) */
    printf("Resolution : %d x %d\n", bmpInfoHeader.biWidth, bmpInfoHeader.biHeight);
    printf("Bit Count : %d(%d:%d)\n", bmpInfoHeader.biBitCount, elemSize, stride);     /* 픽셀당 비트 수(색상) */
    printf("Image Size : %d\n", imageSize);

    inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize); 
    outimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
    fread(inimg, sizeof(ubyte), imageSize, fp); 
    
    fclose(fp);

    //allocate and initialize memory on device
    ubyte *d_inimg = NULL, *d_outimg = NULL;
    hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
    hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);
    hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);
    
    //copy host rgb data array to device rgb data array
    hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

    //define block and grid dimensions
    //const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/4));
    //const dim3 dimBlock(32, 4);
    const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/16), 1);
    const dim3 dimBlock(32, 16, elemSize);
    
    //execute cuda kernel
    convertBlur<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);

    //copy computed blur data array from device to host
    hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

    hipFree(d_outimg);
    hipFree(d_inimg);

    
    if((fp = fopen(argv[2], "wb")) == NULL) { 		/* write bmp */ 
        fprintf(stderr, "Error : Failed to open file...₩n"); 
        return -1;
    }

    /* BITMAPFILEHEADER 구조체의 데이터 */
    fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);

    /* BITMAPINFOHEADER 구조체의 데이터 */
    fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);

    //fwrite(inimg, sizeof(ubyte), imageSize, fp); 
    fwrite(outimg, sizeof(ubyte), imageSize, fp);

    fclose(fp); 
    
    free(inimg); 
    free(outimg);
    
    return 0;
}
