//nvcc     -lcuda

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

// This function wraps the CUDA ttriver API into a template function
template <class T>
inline void getCudaAttribute(T *attribute, hipDeviceAttribute_t device_attribute,
 								int device)
{
	hipError_t error = hipDeviceGetAttribute(attribute, device_attribute, device);
	if(hipSuccess != error) {
		fprintf(stderr, "cuSafeCallNoSync( ) ttriver API error = %04d from file <%s>, line %i.\n", error, __FILE__, __LINE__);
		exit(-1);
	}
}


int main()
{
	printf("CUDA Version %i\n", CUDA_VERSION);
	//printf("CUDA Version %s\n", VERSION_STATUS);
	printf("\nDriver\n");
	
	hipInit(0);
	
	int driverVersion, runtimeVersion;
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf(" CUDA Version : %i\n", driverVersion);
	printf(" Runtime version : %i\n", runtimeVersion);
	
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	printf(" Number of device : %i\n", deviceCount);
	
	int value;
	hipDeviceProp_t prop;

	for(int id = 0; id < deviceCount; id++) {
		hipGetDeviceProperties(&prop, id);
	       	printf(" Device Name : %s\n", prop.name);
	       	printf(" Total global mem : %ld\n", prop.totalGlobalMem);
	       	printf(" Total Constant Mem : %ld\n", prop.totalConstMem);
	       	printf("\n Attribute(%i)\n", id+1);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeMaxThreadsPerBlock, id);
	       	printf(" Max Threads per Block : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeMaxBlockDimX, id);
	       	printf(" Max Block DIM(x) : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeMaxBlockDimY, id);
	       	printf(" Max Block DIM(y) : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeMaxBlockDimZ, id);
	       	printf(" Max Block DIM(z) : %i\n", value);
		getCudaAttribute<int>(&value, hipDeviceAttributeTotalConstantMemory, id);
	       	printf(" Total Constant Memory : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeWarpSize, id); 
	       	printf(" Warp size : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeMemoryClockRate, id);
	       	printf(" Clock Rate : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeMemoryBusWidth, id);
	       	printf(" Memory Bus Width : %i\n", value);
	       	getCudaAttribute<int>(&value, hipDeviceAttributeL2CacheSize, id);
	       	printf(" L2 Cache Size : %i\n", value);
       	}

       	return EXIT_SUCCESS;
}
